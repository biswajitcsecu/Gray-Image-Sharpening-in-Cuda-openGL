#include "hip/hip_runtime.h"
#ifndef GRAPHICSMODE_H
#define GRAPHICSMODE_H
#define GL_H
#define GL_GLEXT_PROTOTYPES
#define GRAPHICS_H
#endif

#define cimg_display 0

#include <cstdio>
#include <cmath>
#include <cstdlib>
#include <GL/gl.h>
#include <GL/freeglut.h>
#include <hip/hip_runtime.h>
#include <cudaGL.h>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_texture_types.h>
#include <>
#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>
#include <CImg.h>




using namespace std;
using namespace cimg_library;

#define REFRESH_DELAY 10 
#define DELTA 10
#define W 512
#define H 512
#define TX 32
#define TY 32
#define RAD 1

static const char *windname = "Image Sharpening";

//Graphics Resource objects
GLuint pbo = 0;
GLuint tex = 0;
struct hipGraphicsResource *cuda_pbo_resource;
struct uchar4;
struct float4;
struct int3 loc = {W/2, H/2, 1};

// Parameters initialization
int sys = 2;
float param = 0.25f;
GLfloat angle1 = 0.0f;
GLfloat angle2 = 0.0f;
float g_fAnim = 0.0;
int mouse_old_x; 
int mouse_old_y;
int mouse_buttons = 0;
float rotate_x = 0.0;
float rotate_y = 0.0;
float translate_z = -3.0;
float scale = 500;








int divUp(int a, int b) { return (a + b - 1) / b; }

__device__
unsigned char clip(int n) { return n > 255 ? 255 : (n < 0 ? 0 : n); }

__device__
int idxClip(int idx, int idxMax) {
    return idx > (idxMax-1) ? (idxMax-1) : (idx < 0 ? 0 : idx);
}

__device__
int flatten(int col, int row, int width, int height) {
    return idxClip(col, width) + idxClip(row, height)*width;
}

__global__
void sharpenKernel(uchar4 *d_out, const uchar4 *d_in, const float *d_filter, int w, int h) {
    const int c = threadIdx.x + blockDim.x * blockIdx.x;
    const int r = threadIdx.y + blockDim.y * blockIdx.y;
    
    if ((c >= w) || (r >= h)) return;
    
    
    const int i = flatten(c, r, w, h);
    const int fltSz = 2*RAD + 1;
    float rgb[3] = {0.f, 0.f, 0.f};
    
    for (int rd = -RAD; rd <= RAD; ++rd) {
        for (int cd = -RAD; cd <= RAD; ++cd) {
            int imgIdx = flatten(c + cd, r + rd, w, h);
            int fltIdx = flatten(RAD + cd, RAD + rd, fltSz, fltSz);
            uchar4 color = d_in[imgIdx];
            float weight = d_filter[fltIdx];
            rgb[0] += weight*color.x;
            rgb[1] += weight*color.y;
            rgb[2] += weight*color.z;
        }
    }
    
    d_out[i].x = clip(rgb[0]);
    d_out[i].y = clip(rgb[1]);
    d_out[i].z = clip(rgb[2]);
}


// Cuda run model
 void cudarun() {
    //Image
    CImg<unsigned char> img ("src2.bmp");
    
    //input data
    uchar4 *arr=(uchar4*)malloc(W*H*sizeof(uchar4));
    
    // Copy data to array
    for (int r = 0; r < H; ++r) {
        for (int c = 0; c < W; ++c){
            arr[r*W + c].x = img(c,r, 0);
            arr[r*W + c].y = img(c,r, 0);
            arr[r*W + c].z = img(c,r, 0);
            arr[r*W + c].w = 0;
        }
    }
    
    //device storage 
    const int fltSz = 2 * RAD + 1;
    const float filter[9] = {1.0/16, 2.0/16, 1.0/16,
			2.0/16, 4.0/16, 2.0/16,
			1.0/16, 2.0/16, 1.0/16};
    
    uchar4 *d_in = 0, *d_out = 0;
    float *d_filter = 0;
    
    hipMalloc(&d_in, W*H*sizeof(uchar4));
    hipMemcpy(d_in, arr, W*H*sizeof(uchar4), hipMemcpyHostToDevice);
    hipMalloc(&d_out, W*H*sizeof(uchar4));
    hipMalloc(&d_filter, fltSz*fltSz*sizeof(float));
    hipMemcpy(d_filter, filter, fltSz*fltSz*sizeof(float),hipMemcpyHostToDevice);
    
       
    //Graphics resources map
    hipGraphicsMapResources(1, &cuda_pbo_resource, 0);
    hipGraphicsResourceGetMappedPointer((void **)&d_out, NULL,cuda_pbo_resource);    
    
    //kernelLauncher   
    const dim3 blockSize(TX, TY);
    const dim3 gridSize(divUp(W, blockSize.x), divUp(H, blockSize.y)); 
    
    //const dim3 blockSize(TX, TY);
    //const dim3 gridSize = dim3((W + TX - 1)/TX, (H + TY - 1)/TY);
    
    sharpenKernel<<<gridSize, blockSize>>>(d_out, d_in, d_filter, W, H);  
    hipDeviceSynchronize();
    hipGraphicsUnmapResources(1, &cuda_pbo_resource, 0);
    
    //SAVE IMAGE: Copy from array to CImg data
    hipMemcpy(arr, d_out, W*H*sizeof(uchar4), hipMemcpyDeviceToHost);
    for (int r =0; r < H; ++r) {
        for (int c= 0; c < W; ++c) {
            img(c,r, 0) = arr[r*W + c].x;
            img(c,r, 1) = arr[r*W + c].y;
            img(c,r, 2) = arr[r*W + c].z;
        }
    }
    img.save_bmp("out.bmp");

 }

 
 
 
// Display model
static void display(){
    
    cudarun();
    glClearColor(0.0,0.34,0.46,1.0);
    glClearDepth(1.0);
    glClear(GL_COLOR_BUFFER_BIT|GL_DEPTH_BUFFER_BIT);
    glEnable(GL_LIGHTING);
    glEnable(GL_DEPTH_TEST);
    glEnable(GL_COLOR_MATERIAL);
    glEnable(GL_LIGHT0);
    glEnable(GL_LIGHT1);
    glShadeModel(GL_SMOOTH);
    glEnable(GL_CULL_FACE);
    glCullFace(GL_BACK);
    glEnable(GL_COLOR_MATERIAL);
    glEnable(GL_NORMALIZE);
    
    
    glPixelStorei(GL_UNPACK_ALIGNMENT, 1);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
    glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, W, H, 0, GL_RGBA,GL_UNSIGNED_BYTE, NULL);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_NEAREST);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);    
    glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
    glTexParameterf(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
    
    //Textue map
    glEnable(GL_TEXTURE_2D);
    glActiveTexture(GL_TEXTURE_2D);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    glBegin(GL_QUADS);
    	glTexCoord2f(0.0f, 0.0f); glVertex2f(0,0);
    	glTexCoord2f(0.0f, 1.0f); glVertex2f(0,H);
    	glTexCoord2f(1.0f, 1.0f); glVertex2f(W,H);
    	glTexCoord2f(1.0f, 0.0f); glVertex2f(W,0);
    glEnd();
    glPopMatrix();
    

    glFlush();
    glDepthFunc(GL_LEQUAL); 
    glHint(GL_PERSPECTIVE_CORRECTION_HINT, GL_NICEST);
    glutSwapBuffers();  
    g_fAnim += 0.05f;
    glDisable(GL_TEXTURE_2D);
    
}


// Reshape window
static void reshape(int w, int h){
    glViewport(0, 0, (GLsizei) w, (GLsizei) h);
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    gluOrtho2D(0, W, H, 0);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();
}

// Pixel Buffer generate
static void initPixelBuffer() {
    glGenBuffers(1, &pbo);
    glBindBuffer(GL_PIXEL_UNPACK_BUFFER, pbo);
    glBufferData(GL_PIXEL_UNPACK_BUFFER, 4*W*H*sizeof(GLubyte), 0, GL_STREAM_DRAW);

    glGenTextures(1, &tex);
    glBindTexture(GL_TEXTURE_2D, tex);
    glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_NEAREST);
    hipGraphicsGLRegisterBuffer(&cuda_pbo_resource, pbo,cudaGraphicsMapFlagsWriteDiscard);
}


// Handler for animation
static void animate(void){
     angle1  = 0.01f*glutGet(GLUT_ELAPSED_TIME);
     angle2 =  0.01f*glutGet(GLUT_ELAPSED_TIME);
     glutPostRedisplay();

}

// Handler for timer event
static void timerEvent(int value){
    if (glutGetWindow()) {
        glutPostRedisplay();
        glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    }
}

// Free Buffer and Texture
static void exitfunc() {
    if (pbo) {
        hipGraphicsUnregisterResource(cuda_pbo_resource);
        glDeleteBuffers(1, &pbo);
        glDeleteTextures(1, &tex);
    }
}

// Handler for mous event
static void mouskey(int button,int state, int x, int y){
    if (state == GLUT_DOWN)    {
        mouse_buttons |= 1<<button;
    }
    else if (state == GLUT_UP)    {
        mouse_buttons = 0;
    }
    mouse_old_x = x;
    mouse_old_y = y;
        
 }


// Handler for key event
static void keyboard(unsigned char key, int x, int y) {
  if(x==0||y==0) return;
    switch (key){
        case (27) :
            if (key==27||key=='q'||key=='Q')
                exit(EXIT_SUCCESS);
            else
                glutDestroyWindow(glutGetWindow());
                return;        
    }
}


static void motion(int x, int y){
    float dx, dy;
    dx = (float)(x - mouse_old_x);
    dy = (float)(y - mouse_old_y);
    if (mouse_buttons & 1){
        rotate_x += dy * 0.2f;
        rotate_y += dx * 0.2f;
    }
    else if (mouse_buttons & 4){
        translate_z += dy * 0.01f;
    }
    mouse_old_x = x;
    mouse_old_y = y;
}


int main(int argc, char** argv) {
    hipDeviceProp_t  prop;
    hipGetDeviceProperties(&prop, 0);
    glutInit(&argc, argv);
    glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE|GLUT_DEPTH);
    glutInitWindowSize(W, H);
    glutInitWindowPosition(200, 200);
    glutCreateWindow(windname);     
    glutDisplayFunc(display);
    glutReshapeFunc(reshape);
    initPixelBuffer();
    glutIdleFunc(animate);
    glutTimerFunc(REFRESH_DELAY, timerEvent,0);
    glutMouseFunc(mouskey);
    glutMotionFunc(motion);
    glutKeyboardFunc(keyboard);
    glutMainLoop();
    atexit(exitfunc);
    return 0;
}
